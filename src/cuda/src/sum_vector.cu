

#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <time.h>
#include <string>


void sumArraysOnHost(float* A, float* B, float* C, int N) {
  for (int idx=0; idx<N; idx++) {
    C[idx] = A[idx] + B[idx];
  }
}

__global__ void checkIndex(void) {
  printf("Hello World from GPU!\n");

}

void initialData(float *ip, int size) {
  time_t t;
  srand((unsigned int) time(&t));
  for (int i = 0; i < size; i++) {
    ip[i] = (float) rand() / (float) RAND_MAX;
  }
}

int main(int argc, char **argv) {
  int nElem = 1024;
  size_t nBytes = nElem * sizeof(float);

  float* h_A;
  float* h_B;
  float* h_C;

  h_A = (float *) malloc(nBytes);
  h_B = (float *) malloc(nBytes);
  h_C = (float *) malloc(nBytes);

  initialData(h_A, nElem);
  initialData(h_B, nElem);

  sumArraysOnHost(h_A, h_B, h_C, nElem);
  free(h_A);
  free(h_B);
  free(h_C);
  return 0;
}